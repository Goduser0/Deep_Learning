#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/entropy_loss_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ForwardGPU(const int nthreads, const Dtype* prob, 
        const Dtype* log_data, const Dtype threshold, const Dtype prob_pow, Dtype* loss_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    if(prob[index] < threshold){
        loss_data[index] = Dtype(0);
    }
    else{
        loss_data[index] = pow(prob[index], prob_pow) * log_data[index];
    }
  }
}

template <typename Dtype>
void EntropyLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* log_data = normalized_bottom_data_.mutable_gpu_data(); 
    
    caffe_gpu_log(data_num_ * label_num_, bottom_data, log_data);

    int nthreads = label_num_ * data_num_;
    Dtype loss;
    Dtype* loss_data = bottom[0]->mutable_gpu_diff();
    ForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads), 
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, bottom_data, log_data, Dtype(0.00001), prob_pow_, loss_data);
    caffe_gpu_asum(label_num_ * data_num_, loss_data, &loss);
    loss = -loss;
    top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
__global__ void EntropyDiff(const int nthreads, const Dtype* data, 
        const Dtype* log_data, const Dtype threshold, 
        const int data_num, const int ignore_label, const int label_num, 
        const Dtype prob_pow, Dtype* count, Dtype* diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    count[index] = Dtype(1) / label_num;
    if(data[index] < threshold){
        diff[index] = Dtype(0);
    }
    else{
        diff[index] = -(
                        pow(data[index], prob_pow - Dtype(1.0)) + 
                        prob_pow * pow(data[index], prob_pow - Dtype(1.0)) * log_data[index]
                       );
    }
  }
}

template <typename Dtype>
void EntropyLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  Dtype* log_data = normalized_bottom_data_.mutable_gpu_data();
  Dtype* count = normalized_bottom_data_.mutable_gpu_diff();
  int nthreads = data_num_ * label_num_;

  const Dtype* bottom_data = bottom[0]->gpu_data();
  if (propagate_down[0]) {
      EntropyDiff<Dtype><<<CAFFE_GET_BLOCKS(nthreads), 
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, bottom_data, log_data, 
                threshold_, data_num_, ignore_label_, label_num_, prob_pow_, count, bottom_diff);

      Dtype count_num;
      caffe_gpu_asum(nthreads, count, &count_num);
      count_num = count_num > 0 ? count_num : Dtype(1);
      caffe_gpu_scal(nthreads, loss_weight_ / count_num, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(EntropyLossLayer);

}  // namespace caffe
